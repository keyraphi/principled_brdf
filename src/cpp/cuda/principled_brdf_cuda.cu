#include "hip/hip_runtime.h"
#include "principled_brdf_cuda.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

// CUDA kernel
__global__ void add_cuda_kernel(const float* a, const float* b, float* c, size_t n) {
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

// CUDA memory allocation
extern "C" void* cuda_allocate(size_t size) {
    void* ptr = nullptr;
    hipError_t err = hipMalloc(&ptr, size);
    if (err != hipSuccess) {
        return nullptr;
    }
    return ptr;
}

// CUDA memory deallocation
extern "C" void cuda_free(void* ptr) {
    if (ptr) {
        hipFree(ptr);
    }
}

// CUDA dummy add implementation
extern "C" void cuda_dummy_add(const float* a, const float* b, float* result, size_t n) {
    // Launch kernel
    add_cuda_kernel<<<(n + 255) / 256, 256>>>(a, b, result, n);
    
    // Synchronize and check for errors
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        // Clean up on error
        hipFree(result);
        throw std::runtime_error("CUDA kernel execution failed");
    }
    
    // Check for kernel launch errors
    err = hipGetLastError();
    if (err != hipSuccess) {
        hipFree(result);
        throw std::runtime_error("CUDA kernel launch failed");
    }
}
